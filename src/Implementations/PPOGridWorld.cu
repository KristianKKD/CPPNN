#include "hip/hip_runtime.h"
#include <shared.hpp>
#include <neuralnetwork.cuh>
#include <library.cuh>
#include <random>

void DrawGrid(const vector<float> grid, int rows, int columns, int agentPos, int loseVal, int winVal) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < columns; ++j) {
            int index = i * columns + j;
            if (index == agentPos)
                std::cout << "0"; // Agent position
            else if (grid[index] == winVal)
                std::cout << "+"; // Win condition
            else if (grid[index] == loseVal)
                std::cout << "X"; // Edge
            else
                std::cout << "-"; // Regular space
        }
        std::cout << std::endl;
    }
}

void GenerateGrid(vector<float>& grid, int rows, int columns, int agentPos, float loseVal, float winVal, float generalVal) {
    int gridSize = rows * columns;
    //generate the base grid
    for (int i = 0; i < gridSize; ++i) {
        if (i <= columns || i >= gridSize - columns || //top and bottom
                i % columns == 0 || (i + 1) % columns == 0) //left and right
            grid[i] = loseVal; //edges are death
        else
            grid[i] = generalVal; //fill the rest of the grid with small negatives to punish lots of moving
    }

    grid[agentPos] = 0; //label agent as '0' to indicate no reward for moving to itself
    grid[gridSize - columns - 2] = winVal; //look for best reward which is placed away from starting pos
}

float RewardFunction(const vector<float> grid, int agentPos, int time, float timeVal, int rows, int columns, float winValue) {
    float reward = 0;
    
    reward += grid[agentPos];
    
    //distance to goal
    int goalPos = grid.size() - columns - 2;
    int goalX = goalPos % columns;
    int goalY = goalPos / columns;
    int agentX = agentPos % columns;
    int agentY = agentPos / columns;
    float distance = abs(goalX - agentX) + abs(goalY - agentY);
    reward += winValue / (winValue + distance);
    
    return reward;
}

void GridWorld() {
    //environment params, generated per iteration
    const int rows = 6;
    const int columns = 6;
    const int gridSize = rows * columns;
    const float loseVal = -50;
    const float winVal = 25;
    const float generalVal = 0;
    const float timeVal = 1;
    const int drawDelay = -100; //draw the grid every n training iterations

    vector<float> grid(gridSize);

    //policy hyper params
    const int pInputs = gridSize; //6x6 cells
    const int pHiddenLayers = 4;
    const int pHiddenSize = 4;
    const int pOutputs = 4; //left, right, up, down
    const int pBatchSize = 5; //iterations between updating the gradients

    //value hyper params
    const int vInputs = gridSize; //6x6 cells
    const int vHiddenLayers = 4;
    const int vHiddenSize = 4;
    const int vOutputs = 1; //estimated reward
    const int vBatchSize = 5; //iterations between updating the gradients

    //greed
    const float greedChanceStart = 0.3;
    float greedChance = greedChanceStart;
    float greedStep = 0.025;

    //learning hyper params
    const int learningIterations = 50000;
    const float learningRate = 0.0001;
    const int timeCutoff = 15; //max steps per try 

    //create policy network
    NeuralNetwork policyNet(pInputs, NeuralNetwork::OutputType::Softmax); //softmax for probability of selection of move
    policyNet.SetActivationFunction(NeuralNetwork::ActivationType::Tanh);
    policyNet.SetGradientClipping(.1);
    policyNet.SetGradientRegularization(0.01);
    policyNet.SetInitMultipliers(0.1, 0.1);
    for (int i = 0; i < pHiddenLayers; i++)
        policyNet.AddLayer(pHiddenSize, true);
    policyNet.AddLayer(pOutputs);
    policyNet.Build();

    //create value network
    NeuralNetwork valueNet(vInputs, NeuralNetwork::OutputType::Raw);
    for (int i = 0; i < vHiddenLayers; i++)
        valueNet.AddLayer(vHiddenSize, false);
    valueNet.SetActivationFunction(NeuralNetwork::ActivationType::Tanh);
    valueNet.SetGradientClipping(.1);
    valueNet.SetGradientRegularization(0.01);
    valueNet.SetInitMultipliers(0.1, 0.1);
    valueNet.AddLayer(vOutputs);
    valueNet.Build();

    //create policy network output arr
    vector<float> pOutputsArr(pOutputs);

    //create value network output arr
    vector<float> vOutputsArr(vOutputs);

    NeuralNetwork oldPolicy = policyNet;

    for (int epoch = 0; epoch < learningIterations; epoch++) {
        if (epoch % pBatchSize == 0)
            policyNet.ApplyGradients(learningRate, pBatchSize);

        int agentPos = columns + 1; //set agent pos to the top left corner next to edges
        
        //create the grid
        GenerateGrid(grid, rows, columns, agentPos, loseVal, winVal, generalVal);

        //draw initial grid
        if (epoch % drawDelay == 0  && drawDelay > 0)
            DrawGrid(grid, rows, columns, agentPos, loseVal, winVal);

        //vecs to save data about the path
        vector<vector<float>> states;
        vector<float> rewards;
        vector<float> chosenProbability;
        vector<int> chosenOptionIndex;

        //reset
        int time = 0;
        greedChance = greedChanceStart; 

        for (; time < timeCutoff; time++) {
            //save current state
            vector<float> state(grid);
            states.push_back(state);

            //normalize input state
            vector<float> normalizedState(state);
            Library::Normalize(normalizedState.data(), gridSize);

            //get probability distribution of moves
            policyNet.FeedForward(normalizedState.data(), pOutputsArr.data());
            valueNet.FeedForward(normalizedState.data(), vOutputsArr.data());

            //select a move based on the probabilities
            int chosenMove = Library::SampleDistribution(pOutputsArr.data(), pOutputs);

            //greedy choice
            if (rand() / float(RAND_MAX) < greedChance)
                chosenMove = rand() % pOutputs;
            greedChance -= greedStep;

            //save this choice
            chosenProbability.push_back(pOutputsArr[chosenMove]);
            chosenOptionIndex.push_back(chosenMove);

            //map the move
            int newPos = 0;
            switch(chosenMove) { 
                case 0: //left
                    newPos = agentPos - 1;
                    break;
                case 1: //right
                    newPos = agentPos + 1;
                    break;
                case 2: //up
                    newPos = agentPos - columns; 
                    break;
                case 3: //down
                    newPos = agentPos + columns;
                    break;
            }
            
            //reset the old pos to nothing
            grid[agentPos] = generalVal;
            agentPos = newPos; //it shouldn't be possible to go out of bounds of the array
            
            //draw for visual representation
            if (epoch % drawDelay == 0 && drawDelay > 0)
                DrawGrid(grid, rows, columns, agentPos, loseVal, winVal);

            //save the reward
            float reward = RewardFunction(grid, newPos, time, timeVal, rows, columns, winVal);
            rewards.push_back(reward);

            //train the value network
            float output = vOutputsArr[0];
            float loss = output - reward;
            valueNet.Backpropagate(&loss);
            if (time % vBatchSize == 0)
                valueNet.ApplyGradients(learningRate, vBatchSize);
            if (epoch % 200 == 0 && (grid[newPos] == loseVal || grid[newPos] == winVal))
                Log("Epoch:" + to_string(epoch) + "/" + to_string(learningIterations) +
                ", Time:" + to_string(time) + "/" + to_string(timeCutoff) + ", ValueLoss:" + to_string(loss) +
                ", PolicyReward:" + to_string(Library::SumVector(rewards.data(), rewards.size())));

            //find out if agent crashed into edge or touched the win
            if (grid[newPos] == loseVal || grid[newPos] == winVal)
                break;
        }
        
        //calculate cumulative loss
        vector<float> loss(pOutputs, 0);
        for (int i = 0; i < time; i++) {
            //normalize input
            vector<float> normalizedState(states[i]);
            Library::Normalize(normalizedState.data(), gridSize);

            //get the predicted loss
            valueNet.FeedForward(normalizedState.data(), vOutputsArr.data());

            //calculate advantage
            float predicatedValue = vOutputsArr[0];
            float advantage = rewards[i] - predicatedValue;

            //compare old policy prediction
            oldPolicy.FeedForward(normalizedState.data(), pOutputsArr.data());
            float oldProbability = pOutputsArr[chosenOptionIndex[i]];
            float newProbability = chosenProbability[i];
            float ratio = newProbability/oldProbability;

            float clippedLoss = std::min(ratio * advantage, std::clamp(ratio, 1- learningRate, 1 + learningRate) * advantage);
            loss[chosenOptionIndex[i]] += clippedLoss;
        }

        //average loss
        if (time > 0) {
            for (int i = 0; i < pOutputs; i++)
                loss[i] = ((loss[i] != 0) ? (loss[i] / time) : 0);

            policyNet.Backpropagate(loss.data());
        }

        if (epoch % pBatchSize == 0)
            policyNet.ApplyGradients(learningRate, pBatchSize);

        //save this policy as the old policy
        if (time > 0)
            oldPolicy = policyNet;
    }


    //watch it do what it has learnt
    int agentPos = columns + 1; //set agent pos to the top left corner next to edges
    
    //create the grid
    GenerateGrid(grid, rows, columns, agentPos, loseVal, winVal, generalVal);

    //draw initial grid
    DrawGrid(grid, rows, columns, agentPos, loseVal, winVal);

    for (int time = 0; time < timeCutoff; time++) {
        //save current state
        vector<float> state(grid);

        vector<float> normalizedState(state);
        Library::Normalize(normalizedState.data(), gridSize);

        //get probability distribution of moves
        policyNet.FeedForward(normalizedState.data(), pOutputsArr.data());

        //select a move based on the probabilities
        int chosenMove = Library::SampleDistribution(pOutputsArr.data(), pOutputs);

        //map the move
        int newPos = 0;
        switch(chosenMove) { 
            case 0: //left
                newPos = agentPos - 1;
                break;
            case 1: //right
                newPos = agentPos + 1;
                break;
            case 2: //up
                newPos = agentPos - columns; 
                break;
            case 3: //down
                newPos = agentPos + columns;
                break;
        }
        
        //reset the old pos to nothing
        grid[agentPos] = generalVal;
        agentPos = newPos; //it shouldn't be possible to go out of bounds of the array
        
        //draw for visual representation
        DrawGrid(grid, rows, columns, agentPos, loseVal, winVal);

        //find out if agent crashed into edge or touched the win
        if (grid[newPos] == loseVal || grid[newPos] == winVal)
            break;
    }


    return;
}