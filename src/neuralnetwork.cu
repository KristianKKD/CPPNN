#include "hip/hip_runtime.h"
#include "library.cuh"
#include "neuralnetwork.cuh"
  
#define CUDACHECK(call) {                                                        \
        hipError_t err = call;                                                  \
        if (err != hipSuccess) {                                                \
            std::cerr << "CUDA Error: " << hipGetErrorString(err) << " at "     \
                      << __FILE__ << ":" << __LINE__ << std::endl;               \
            exit(EXIT_FAILURE);                                                  \
        }                                                                        \
    }

NeuralNetwork::NeuralNetwork(int inputSize) {
    std::fill(this->layerSizes, this->layerSizes + LIMITLAYERCOUNT, 0); //tracking
    std::fill(this->normLayer, this->normLayer + LIMITLAYERCOUNT, false); //bool mask for normalization
    std::fill(this->scales, this->scales + LIMITLAYERCOUNT, 1); //* 1
    std::fill(this->shifts, this->shifts + LIMITLAYERCOUNT, 0); //+ 0
    //weights/biases don't matter because we will change them anyway

    this->weightCount = 0;
    this->nodeCount = 0;
    this->layerCount = 0;
    this->AddLayer(inputSize);
}

NeuralNetwork::~NeuralNetwork() {
    hipFree(this->weights);
    hipFree(this->biases);
    hipFree(this->activatedOutputs);
}

void NeuralNetwork::AddLayer(int size, bool normalized = false) {
    int lastLayerSize = 0;
    if (this->layerCount > 0)
        lastLayerSize = this->layerSizes[this->layerCount - 1];

    int newWeightCount = lastLayerSize * size;
    this->weightCount += newWeightCount;

    this->layerSizes[this->layerCount] = size;
    this->normLayer[this->layerCount] = normalized;
    this->layerCount++;
    this->nodeCount += size;
}

__global__ void Sum(float* activatedOutputs, const float* weights, 
                    const int layerSize, const int nextLayerSize,
                    const long long nodeOffset, const long long weightOffset) {
                        
    //weights work as follows in example:
    //3 input size (0,1,2), 2 hidden size (3,4) 
    //0->3 = E0, 1->3 = E1, 2->3 = E2
    //0->4 = E3, 1->3 = E4, 2->3 = E5

    int i = blockIdx.x * blockDim.x + threadIdx.x; //this might cause an error if there aren't enough blocks/threads?

    if (i > layerSize * nextLayerSize - 1)
        return;

    long long targetIn = (i % layerSize) + nodeOffset; //0, 1, 2, 0, 1, 2
    long long targetWeight = i + weightOffset; //0, 1, 2, 3, 4, 5
    long long targetNode = layerSize + (i / layerSize) + nodeOffset; //3, 3, 3, 4, 4, 4 
    
    float outputVal = activatedOutputs[targetIn] * weights[targetWeight];

    atomicAdd(&activatedOutputs[targetNode], outputVal);

    // printf("Thread %d:      IN[%d]:%f      W[%d]:%f      OUT[%d]:%f     VAL:%f\n", 
    //         i, 
    //         targetIn, activatedOutputs[targetIn],
    //         targetWeight, weights[targetWeight],
    //         targetNode, activatedOutputs[targetNode],
    //         outputVal);
}

__global__ void ActivateLayer(float* activatedOutputs, const int layerSize, const long long nodeOffset) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= layerSize)
        return;

    //printf("Activating index %d from value %f\n", nodeOffset + i, activatedOutputs[nodeOffset + i]);
    Library::ActivationFunction(&activatedOutputs[nodeOffset + i]);
}

void NeuralNetwork::Build() {
    if (this->layerCount > LIMITLAYERCOUNT)
        throw std::runtime_error("Too many layers!");

    //initialize cuda device
    hipSetDevice(Library::gpuDevice); //maybe pointless?

    //memory allocation
    CUDACHECK(hipMallocManaged(&this->weights, this->weightCount * sizeof(float))); //weights
    CUDACHECK(hipMallocManaged(&this->biases, (this->nodeCount - this->layerSizes[0]) * sizeof(float))); //biases
    CUDACHECK(hipMallocManaged(&this->activatedOutputs, this->nodeCount * sizeof(float))); //node outputs

    //randomly initialize weights
    for (long long i = 0; i < this->weightCount; i++) {
        float rand = Library::RandomValue();
        if (rand > 0.5)
            rand = 1;
        else
            rand = 0.1;
        this->weights[i] = rand;
    }

    //randomly initialize biases
    for (int i = 0; i < this->nodeCount - this->layerSizes[0]; i++) { //no bias for input layer
        // float rand = Library::RandomValue();
        // if (rand > 0.5)
        //     rand = 1;
        // else
        //     rand = 0.1;
        // this->biases[i] = rand;
        this->biases[i] = 0;
    }

    for (int i = 0; i < this->layerCount; i++){
        this->shifts[i] = 0;
        this->scales[i] = 1;
    }

    // //prefetch the data we know we will use soon for some small performance boost
    // CUDACHECK(hipMemPrefetchAsync(this->activatedOutputs, this->nodeCount * sizeof(float), Library::gpuDevice));
    // CUDACHECK(hipMemPrefetchAsync(this->weights, this->weightCount * sizeof(float), Library::gpuDevice));
    // CUDACHECK(hipMemPrefetchAsync(this->biases, (this->nodeCount - this->layerSizes[0]) * sizeof(float), Library::gpuDevice));

    //some calculations to save time later
    int largestLayerSize = 0;
    int largestLayerWeightCount = 0;
    for (int i = 1; i < this->layerCount; i++) {
        int layerSize = this->layerSizes[i];
        if (layerSize > largestLayerSize)
            largestLayerSize = layerSize;

        int weightConnections = layerSize * this->layerSizes[i - 1];
        if (weightConnections > largestLayerWeightCount)
            largestLayerWeightCount = weightConnections;
    }
    this->largestLayerSize = largestLayerSize;
    this->largestLayerWeightCount = largestLayerWeightCount;

    CUDACHECK(hipDeviceSynchronize()); //finish operations
}

void NeuralNetwork::FeedForward(float* inputArr, float* outputArr) {
    //get stats for CUDA so we don't go out of bounds
    hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, Library::gpuDevice);
    
    //calculate the sizes of the CUDA blocks
    int inputBlocksNeeded = min((this->layerSizes[0] + THREADSPERBLOCK - 1) / THREADSPERBLOCK, properties.maxGridSize[0]);
    int sumBlocksNeeded = min((this->largestLayerWeightCount + THREADSPERBLOCK - 1) / THREADSPERBLOCK, properties.maxGridSize[0]);
    int activationBlocksNeeded = min((this->largestLayerSize + THREADSPERBLOCK - 1) / THREADSPERBLOCK, properties.maxGridSize[0]);

    //copy the input into the outputs array, fill other slots with biases
    CUDACHECK(hipMemcpy(this->activatedOutputs, inputArr, this->layerSizes[0] * sizeof(float), hipMemcpyHostToDevice));
    CUDACHECK(hipMemcpy(this->activatedOutputs + this->layerSizes[0], this->biases, (this->nodeCount - this->layerSizes[0]) * sizeof(float), hipMemcpyHostToDevice));
    CUDACHECK(hipDeviceSynchronize());

    //activate the input layer
    ActivateLayer<<<inputBlocksNeeded, THREADSPERBLOCK>>>(this->activatedOutputs, this->layerSizes[0], 0); 
    CUDACHECK(hipDeviceSynchronize());

    //iterate over the nodes, saving all of their values into activatedOutputs, fn(sum * weights) + bias to calculate the next layer outs
    long long usedNodes = 0;
    long long usedWeights = 0;
    for (int layerIndex = 0; layerIndex < this->layerCount - 1; layerIndex++) {
        int layerSize = this->layerSizes[layerIndex];
        int nextLayerSize = this->layerSizes[layerIndex + 1]; 
        
        //sum all the outputs * weights
        Sum<<<sumBlocksNeeded, THREADSPERBLOCK>>>(this->activatedOutputs, this->weights, layerSize, nextLayerSize, usedNodes, usedWeights); 
        CUDACHECK(hipDeviceSynchronize());

        //indexing shortcut shenanigans
        usedNodes += layerSize;
        usedWeights += layerSize * nextLayerSize;

        //normalization layer
        if (this->normLayer[layerIndex]) { //TODO, IMPLEMENT NORMALIZATION ON GPU?
            //calc mean
            float sum = 0;
            for (int sumIndex = 0; sumIndex < nextLayerSize; sumIndex++)
                sum += this->activatedOutputs[usedNodes + sumIndex];
            if (sum == 0)
                sum += EPSILON;
            float mean = sum/(float)nextLayerSize;

            //calc std
            float variance = 0;
            for (int varIndex = 0; varIndex < nextLayerSize; varIndex++) {
                float val = this->activatedOutputs[usedNodes + varIndex];
                variance += (val - mean) * (val - mean);
            }
            if (variance == 0)
                variance += EPSILON;
            float std = std::sqrt(variance/(float)nextLayerSize);

            for (int normIndex = 0; normIndex < nextLayerSize; normIndex++) {
                float val = this->activatedOutputs[usedNodes + normIndex];
                float newVal = val - mean;
                if (newVal == 0)
                    newVal += EPSILON;
                
                newVal = this->scales[layerIndex] * (newVal/std) + this->shifts[layerIndex];
                this->activatedOutputs[usedNodes + normIndex] = newVal;
                //Log("Normalized output " + to_string(usedNodes + normIndex) + " from " + to_string(val) + " -> " + to_string(newVal));
            }

            CUDACHECK(hipDeviceSynchronize());
        }   

        //activate the next layer's outputs
        ActivateLayer<<<activationBlocksNeeded, THREADSPERBLOCK>>>(this->activatedOutputs, nextLayerSize, usedNodes); 
        CUDACHECK(hipDeviceSynchronize());
    }

    //output by copying the contents of the nodes in the output layer into the arr
    int outputSize = this->layerSizes[this->layerCount - 1];
    for (int i = 0; i < outputSize; i++) {
        float val = this->activatedOutputs[this->nodeCount - outputSize + i];
        outputArr[i] = val;
    }
}

void NeuralNetwork::PrintNetwork() {
    int seenNodes = 0;
    int seenWeights = 0;
    for (int layerIndex = 0; layerIndex < this->layerCount; layerIndex++) {
        int layerSize = this->layerSizes[layerIndex];

        for (int nodeIndex = 0; nodeIndex < layerSize; nodeIndex++) {
            Log("N" + to_string(seenNodes) + " - " + to_string(this->activatedOutputs[seenNodes]));
            
            if (this->layerCount > layerIndex + 1)
                for (int edgeIndex = 0; edgeIndex < this->layerSizes[layerIndex + 1]; edgeIndex++)
                    Log("   E" + to_string(nodeIndex + (edgeIndex * layerSize) + seenWeights) + " - " + to_string(this->weights[nodeIndex + (edgeIndex * layerSize) + seenWeights]));

            seenNodes++;
        } //nodes

        if (this->layerCount > layerIndex + 1)
            seenWeights += layerSize * this->layerSizes[layerIndex + 1];
    } //layers
}

void NeuralNetwork::SetWeights(const float* hostWeights) {
    CUDACHECK(hipMemcpy(this->weights, hostWeights, this->weightCount * sizeof(float), hipMemcpyHostToDevice));
    CUDACHECK(hipDeviceSynchronize());
}

void NeuralNetwork::SetBiases(const float* hostBiases) {
    CUDACHECK(hipMemcpy(this->biases, hostBiases, (this->nodeCount - this->layerSizes[0]) * sizeof(float), hipMemcpyHostToDevice));
    CUDACHECK(hipDeviceSynchronize());
}

void NeuralNetwork::GradientDescent(int changeCount) {
    //make changeCount changes to a random weight
    for (int i = 0; i < changeCount; i++) {
        long long randIndex = std::round(Library::RandomValue() * (this->weightCount - 1));

        float randChange = Library::RandomValue();
        float randDir = (Library::RandomValue() > 0.5) ? 1 :  -1;
        randChange *= randDir;

        float* val = new float;
        *val = this->weights[randIndex] + randChange;
        CUDACHECK(hipMemcpy(this->weights + randIndex, val, sizeof(float), hipMemcpyHostToDevice));
        delete val;
    }
    CUDACHECK(hipDeviceSynchronize());
}