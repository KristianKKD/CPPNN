#include "hip/hip_runtime.h"
#include "library.cuh"
#include "neuralnetwork.cuh"
  
#define CUDACHECK(call) {                                                        \
        hipError_t err = call;                                                  \
        if (err != hipSuccess) {                                                \
            std::cerr << "CUDA Error: " << hipGetErrorString(err) << " at "     \
                      << __FILE__ << ":" << __LINE__ << std::endl;               \
            exit(EXIT_FAILURE);                                                  \
        }                                                                        \
    }

NeuralNetwork::NeuralNetwork(int inputSize) {
    this->weightCount = 0;
    this->nodeCount = 0;
    this->layerCount = 0;
    this->AddLayer(inputSize);
}

NeuralNetwork::~NeuralNetwork() {
    hipFree(this->weights);
    hipFree(this->activatedOutputs);
}

void NeuralNetwork::AddLayer(int size) {
    int lastLayerSize = 0;
    if (this->layerCount > 0)
        lastLayerSize = this->layerSizes[this->layerCount - 1];

    int newWeightCount = lastLayerSize * size;
    this->weightCount += newWeightCount;

    this->layerSizes[this->layerCount] = size;
    this->layerCount++;
    this->nodeCount += size;
}

__global__ void Sum(float* activatedOutputs, const float* weights, 
                    const int layerSize, const int nextLayerSize,
                    const long long nodeOffset, const long long weightOffset) {
                        
    //weights work as follows in example:
    //3 input size (0,1,2), 2 hidden size (3,4) 
    //0->3 = E0, 1->3 = E1, 2->3 = E2
    //0->4 = E3, 1->3 = E4, 2->3 = E5

    int i = blockIdx.x * blockDim.x + threadIdx.x; //this might cause an error if there aren't enough blocks/threads?

    if (i > layerSize * nextLayerSize - 1)
        return;

    long long targetIn = (i % layerSize) + nodeOffset; //0, 1, 2, 0, 1, 2
    long long targetWeight = i + weightOffset; //0, 1, 2, 3, 4, 5
    long long targetNode = layerSize + (i / layerSize) + nodeOffset; //3, 3, 3, 4, 4, 4 
    
    float outputVal = activatedOutputs[targetIn] * weights[targetWeight];

    atomicAdd(&activatedOutputs[targetNode], outputVal);

    // printf("Thread %d:      IN[%d]:%f      W[%d]:%f      OUT[%d]:%f     VAL:%f\n", 
    //         i, 
    //         targetIn, activatedOutputs[targetIn],
    //         targetWeight, weights[targetWeight],
    //         targetNode, activatedOutputs[targetNode],
    //         outputVal);
}

__global__ void ActivateLayer(float* activatedOutputs, const int layerSize, const long long nodeOffset) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= layerSize)
        return;

    //printf("Activating index %d from value %f\n", nodeOffset + i, activatedOutputs[nodeOffset + i]);
    Library::ActivationFunction(&activatedOutputs[nodeOffset + i]);
}

void NeuralNetwork::Build() {
    if (this->layerCount > LIMITLAYERCOUNT)
        throw std::runtime_error("Too many layers!");

    //initialize cuda device
    hipSetDevice(Library::gpuDevice); //maybe pointless?

    //memory allocation
    CUDACHECK(hipMallocManaged(&this->weights, this->weightCount * sizeof(float))); //weights
    CUDACHECK(hipMallocManaged(&this->biases, (this->nodeCount - this->layerSizes[0]) * sizeof(float))); //biases
    CUDACHECK(hipMallocManaged(&this->activatedOutputs, this->nodeCount * sizeof(float))); //node outputs

    //randomly initialize weights
    for (int i = 0; i < this->weightCount; i++) {
        float rand = Library::RandomValue();
        if (rand > 0.5)
            rand = 1;
        else
            rand = 0.1;
        this->weights[i] = rand;
    }

    //randomly initialize biases
    for (int i = 0; i < this->nodeCount - this->layerSizes[0]; i++) { //no bias for input layer
        // float rand = Library::RandomValue();
        // if (rand > 0.5)
        //     rand = 1;
        // else
        //     rand = 0.1;
        // this->biases[i] = rand;
        this->biases[i] = 0;
    }

    for (int i = 0; i < this->layerCount; i++){
        this->shifts[i] = 0;
        this->scales[i] = 1;
    }

    // //prefetch the data we know we will use soon for some small performance boost
    // CUDACHECK(hipMemPrefetchAsync(this->activatedOutputs, this->nodeCount * sizeof(float), Library::gpuDevice));
    // CUDACHECK(hipMemPrefetchAsync(this->weights, this->weightCount * sizeof(float), Library::gpuDevice));
    // CUDACHECK(hipMemPrefetchAsync(this->biases, (this->nodeCount - this->layerSizes[0]) * sizeof(float), Library::gpuDevice));

    //some calculations to save time later
    int largestLayerSize = 0;
    int largestLayerWeightCount = 0;
    for (int i = 1; i < this->layerCount; i++) {
        int layerSize = this->layerSizes[i];
        if (layerSize > largestLayerSize)
            largestLayerSize = layerSize;

        int weightCount = layerSize * this->layerSizes[i - 1];
        if (weightCount > largestLayerWeightCount)
            largestLayerWeightCount = weightCount;
    }
    this->largestLayerSize = largestLayerSize;
    this->largestLayerWeightCount = largestLayerWeightCount;

    CUDACHECK(hipDeviceSynchronize()); //finish operations
}

void NeuralNetwork::FeedForward(float* inputArr, float* outputArr) {
    //get stats for CUDA so we don't go out of bounds
    hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, Library::gpuDevice);
    
    //calculate the sizes of the CUDA blocks
    int inputBlocksNeeded = min((this->layerSizes[0] + THREADSPERBLOCK - 1) / THREADSPERBLOCK, properties.maxGridSize[0]);
    int sumBlocksNeeded = min((this->largestLayerWeightCount + THREADSPERBLOCK - 1) / THREADSPERBLOCK, properties.maxGridSize[0]);
    int activationBlocksNeeded = min((this->largestLayerSize + THREADSPERBLOCK - 1) / THREADSPERBLOCK, properties.maxGridSize[0]);

    //copy the input into the outputs array, fill other slots with biases
    CUDACHECK(hipMemcpy(this->activatedOutputs, inputArr, this->layerSizes[0] * sizeof(float), hipMemcpyHostToDevice));
    CUDACHECK(hipMemcpy(this->activatedOutputs + this->layerSizes[0], this->biases, (this->nodeCount - this->layerSizes[0]) * sizeof(float), hipMemcpyHostToDevice));
    CUDACHECK(hipDeviceSynchronize());

    //activate the input layer
    ActivateLayer<<<inputBlocksNeeded, THREADSPERBLOCK>>>(this->activatedOutputs, this->layerSizes[0], 0); 
    CUDACHECK(hipDeviceSynchronize());

    //iterate over the nodes, saving all of their values into activatedOutputs, fn(sum * weights) + bias to calculate the next layer outs
    long long usedNodes = 0;
    long long usedWeights = 0;
    int normalizationLayersUsed = 0;
    for (int layerIndex = 0; layerIndex < this->layerCount - 1; layerIndex++) {
        int layerSize = this->layerSizes[layerIndex];
        int nextLayerSize = this->layerSizes[layerIndex + 1]; 
        
        //sum all the outputs * weights
        Sum<<<sumBlocksNeeded, THREADSPERBLOCK>>>(this->activatedOutputs, this->weights, layerSize, nextLayerSize, usedNodes, usedWeights); 
        CUDACHECK(hipDeviceSynchronize());

        //indexing shortcut shenanigans
        usedNodes += layerSize;
        usedWeights += layerSize * nextLayerSize;

        //normalization layer
        if (false) { //TODO, IMPLEMENT NORMALIZATION ON GPU?
            //calc mean
            float sum = 0;
            for (int i = 0; i < nextLayerSize; i++)
                sum += this->activatedOutputs[usedNodes + i];
            if (sum == 0)
                sum += EPSILON;
            float mean = sum/(float)nextLayerSize;

            //calc std
            float variance = 0;
            for (int i = 0; i < nextLayerSize; i++) {
                float val = this->activatedOutputs[usedNodes + i];
                variance += (val - mean) * (val - mean);
            }
            if (variance == 0)
                variance += EPSILON;
            float std = std::sqrt(variance/(float)nextLayerSize);

            for (int i = 0; i < nextLayerSize; i++) {
                float val = this->activatedOutputs[usedNodes + i];
                float newVal = val - mean;
                if (newVal == 0)
                    newVal += EPSILON;
                
                newVal = this->scales[normalizationLayersUsed] * (newVal/std) + this->shifts[normalizationLayersUsed];
                this->activatedOutputs[usedNodes + i] = newVal;
                Log("Normalized output " + to_string(usedNodes + i) + " from " + to_string(val) + " -> " + to_string(newVal));
            }

            normalizationLayersUsed++;
            CUDACHECK(hipDeviceSynchronize());
        }   

        //activate the next layer's outputs
        ActivateLayer<<<activationBlocksNeeded, THREADSPERBLOCK>>>(this->activatedOutputs, nextLayerSize, usedNodes); 
        CUDACHECK(hipDeviceSynchronize());
    }

    //output by copying the contents of the nodes in the output layer into the arr
    int outputSize = this->layerSizes[this->layerCount - 1];
    for (int i = 0; i < outputSize; i++) {
        float val = this->activatedOutputs[this->nodeCount - outputSize + i];
        outputArr[i] = val;
    }
}

void NeuralNetwork::PrintNetwork() {
    int seenNodes = 0;
    int seenWeights = 0;
    for (int layerIndex = 0; layerIndex < this->layerCount; layerIndex++) {
        int layerSize = this->layerSizes[layerIndex];

        for (int nodeIndex = 0; nodeIndex < layerSize; nodeIndex++) {
            Log("N" + to_string(seenNodes) + " - " + to_string(this->activatedOutputs[seenNodes]));
            
            if (this->layerCount > layerIndex + 1)
                for (int edgeIndex = 0; edgeIndex < this->layerSizes[layerIndex + 1]; edgeIndex++)
                    Log("   E" + to_string(nodeIndex + (edgeIndex * layerSize) + seenWeights) + " - " + to_string(this->weights[nodeIndex + (edgeIndex * layerSize) + seenWeights]));

            seenNodes++;
        } //nodes

        if (this->layerCount > layerIndex + 1)
            seenWeights += layerSize * this->layerSizes[layerIndex + 1];
    } //layers
}

void NeuralNetwork::SetWeights(const float* hostWeights) {
    CUDACHECK(hipMemcpy(this->weights, hostWeights, this->weightCount * sizeof(float), hipMemcpyHostToDevice));
    CUDACHECK(hipDeviceSynchronize());
}

void NeuralNetwork::SetBiases(const float* hostBiases) {
    CUDACHECK(hipMemcpy(this->biases, hostBiases, (this->nodeCount - this->layerSizes[0]) * sizeof(float), hipMemcpyHostToDevice));
    CUDACHECK(hipDeviceSynchronize());
}

void NeuralNetwork::GradientDescent(int changeCount) {
    //make changeCount changes to a random weight
    for (int i = 0; i < changeCount; i++) {
        int randIndex = std::round(Library::RandomValue() * (this->weightCount - 1));

        float randChange = Library::RandomValue();
        float randDir = (Library::RandomValue() > 0.5) ? 1 :  -1;
        randChange *= randDir;

        float* val = new float;
        *val = this->weights[randIndex] + randChange;
        CUDACHECK(hipMemcpy(this->weights + randIndex, val, sizeof(float), hipMemcpyHostToDevice));
        delete val;
    }
    CUDACHECK(hipDeviceSynchronize());
}